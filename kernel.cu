#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

//checking error di cuda
void checkCUDAError(const char* msg) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		fprintf(stderr, "cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

//fungsi kernel untuk dieksekusi di GPU
__global__ void kernel(uchar4* pos, unsigned int w, unsigned int h, float time) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x; //pembagian array berdasarkan index array dan index threadnya
	unsigned int x = idx%w, y = idx%w;
	if (idx < w*h) {
		unsigned char r = (x + (int)time) & 0xff;
		unsigned char g = (y + (int)time) & 0xff;
		unsigned char b = (x + y + (int)time) & 0xff;

		//thread menulis lokasi piksel di texture
		pos[idx].w = 0;
		pos[idx].x = r;
		pos[idx].y = g;
		pos[idx].z = b;
	}
}

//wrapper for global call
extern "C" void launch_kernel(uchar4* pos, unsigned int iw, unsigned int ih, float time) {
	//execute kernel
	int nThreads = 256;
	int total = iw*ih; //jumlah data yang akan diproses oleh GPU
	int nBlocks = total / nThreads; //jumlah block yang akan digunakan
	nBlocks += (total%nThreads > 0 ? 0 : 1);

	kernel << <nBlocks, nThreads >> > (pos, iw, ih, time); //pemanggilan fungsi kernel
	hipDeviceSynchronize(); //penyelarasan thread untuk memastikan thread sudah selesai semua
	checkCUDAError("kernel failed");
}